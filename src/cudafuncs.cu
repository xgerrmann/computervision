#include "hip/hip_runtime.h"
//cudafuncs.cu

#include "cudafuncs.hpp"

static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number)
{
	if(err!=hipSuccess)
	{
		fprintf(stderr,"%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n",msg,file_name,line_number,hipGetErrorString(err));
		std::cin.get();
		exit(EXIT_FAILURE);
	}
}

__global__ void calcmap_cuda(int *xp_c, int *yp_c, int *wp_c, float *mxp_c, float *myp_c, float *h_c, int *width, int *height){
	// TODO: max number of blocks
	//int cuda_index = blockDim.x*blockIdx.x + threadIdx.x;
	int c = blockIdx.x*blockDim.x + threadIdx.x;
	int r = blockIdx.y*blockDim.y + threadIdx.y;
	// Check if within image bounds
	if((c>=(*width))||(r>=(*height))) return;
	int cuda_index = r*(*width)+c;
	// First calculate the scale, for the X and Y must be devicd by the scale.
	float w				= (h_c[2]*xp_c[cuda_index]+h_c[5]*yp_c[cuda_index]+h_c[8]*wp_c[cuda_index]);
	// x/w
	mxp_c[cuda_index]	= (h_c[0]*xp_c[cuda_index]+h_c[3]*yp_c[cuda_index]+h_c[6]*wp_c[cuda_index])/w;
	// y/w
	myp_c[cuda_index]	= (h_c[1]*xp_c[cuda_index]+h_c[4]*yp_c[cuda_index]+h_c[7]*wp_c[cuda_index])/w;
}
//
//__global__ void domap_cuda(uchar *image_out, uchar *image_in, float *xp_c, float *yp_c, int *width, int *height){
//	int c = blockIdx.x*blockDim.x + threadIdx.x;
//	int r = blockIdx.y*blockDim.y + threadIdx.y;
//	// Check if within image bounds
//	//if((c>=(*width))||(r>=(*height))) return;
//	int cuda_index = r*(*width)+c;
//	//image_out[cuda_index] = image_in[cuda_index];
//	image_out[cuda_index] = 90;
//}

__global__ void copy_cuda(unsigned char* input,
							unsigned char* output,
							int width,
							int height,
							int step_in,
							int step_out)
{
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
	if((xIndex<width) && (yIndex<height)){
		//const int index = yIndex*step + (3*xIndex);
		const int index			= yIndex*step_in	+ (3*xIndex);
		//const int index_out			= yIndex*outputStep	+ (3*xIndex);
		//const int index_out	= yIndex*step_out	+ (3*xIndex);
		//const int index		= xIndex*step + yIndex;
		output[index]	= input[index];
		output[index+1]	= input[index+1];
		output[index+2]	= input[index+2];
	}
	// no return
}

// Partial wrapper for the __global__ calls
void calcmapping(Eigen::MatrixXf *Mx, Eigen::MatrixXf *My,  Eigen::Matrix3f *Hi, int xmin_out, int ymin_out, int wmax, int hmax){
	//###if(_CUDAFUNCS_DEBUG)
	//##std::cerr << "### calcmapping <start> ###" << std::endl;
	//###endif
	//##// Get the properties of the GPU device, this will only be executed once.
	//##static hipDeviceProp_t cuda_properties;
	//##static hipError_t cuda_error= hipGetDeviceProperties(&cuda_properties,0); // cuda properties of device 0
	//##static int N_BLOCKS_MAX		= cuda_properties.maxThreadsPerBlock;	// x dimension
	//##static int N_THREADS_MAX	= cuda_properties.maxGridSize[0];		// x dimension
	//##static int N_PIXELS_MAX = N_BLOCKS_MAX * N_THREADS_MAX;
	//###if(_CUDAFUNCS_DEBUG)
	//##std::cerr << "N_BLOCKS_MAX: " << N_BLOCKS_MAX << std::endl;
	//##std::cerr << "N_THREADS_MAX:" << N_THREADS_MAX << std::endl;
	//###endif
	//###if(_CUDAFUNCS_TIMEIT)
	//##gputimer watch;
	//##watch.start();
	//###endif

	//##//std::cerr << "Enter calcmapping." << std::endl;
	//##// Calculate max x and y of image
	//##int xmax,ymax;
	//##xmax = xmin_out + wmax - 1;
	//##ymax = ymin_out + hmax - 1;

	//##// Prepare inputs for the device code
	//##// STATIC because every loop this is the same
	//##// Input are meshgrid MATLAB-like arrays of the X and Y coordinates of the pixels and the scale (=1)
	//##arma::Mat<int> x = arma::linspace<arma::Row<int> >(xmin_out,xmax,wmax);
	//##arma::Mat<int> X = arma::repmat(x,hmax,1);
	//##arma::Mat<int> y = arma::linspace<arma::Col<int> >(ymin_out,ymax,hmax);
	//##arma::Mat<int> Y = arma::repmat(y,1,wmax);
	//##arma::Mat<int> W = arma::ones<arma::Mat<int> >(hmax,wmax);
	//##
	//###if(_CUDAFUNCS_DEBUG)
	//##//X.print("X:");
	//##//Y.print("Y:");
	//##//W.print("W:");
	//###endif
	//##
	//##// Determine data sizes
	//##int N		= hmax*wmax;
	//##//std::cerr << hmax << "," << wmax << std::endl;
	//##assert(N<N_PIXELS_MAX);// number of pixels must be smaller then the total number of threads (in the x dimension)
	//##int size_i	= N*sizeof(int);
	//##int size_f	= N*sizeof(float);
	//##int size_h	= 9*sizeof(float); // H (in fact a 3x3 matrix) contains 9 float scalars.

	//##// determine number of blocks and threads per block
	//##//int n_blocks	= ceil(float(N)/float(N_THREADS_MAX));
	//##//int n_threads	= ceil(float(N)/float(n_blocks));
	//##//int n_threads	= N_THREADS_MAX;
	//##//std::cerr << "n_blocks:  "<< n_blocks << std::endl;
	//##//std::cerr << "n_threads: "<< n_threads << std::endl;

	//##// Create pointers to host and device data
	//##int		*xp, *yp, *wp, *xp_c, *yp_c, *wp_c;
	//##float	*mxp, *myp, *hp, *mxp_c, *myp_c, *h_c;
	//##
	//##// Link the pointers to the corresponding data
	//##xp = X.memptr(); // pointer to x matrix input data
	//##yp = Y.memptr(); // pointer to y matrix input data
	//##wp = W.memptr(); // pointer to w matrix input data
	//##hp = Hi->data(); // Hi is a pointer to an eigen matrix
	//##
	//##// Number of rows and columns in Mx and My must be identical
	//##// TODO: Actually this does not have to be the case!!
	//##assert(Mx->rows() == My->rows() && Mx->cols() == My->cols());
	//##// Get pointers to data of mapping matrices
	//##mxp = Mx->data();	// Mx is a pointer, thus child accessing with ->
	//##myp = My->data();	// My is a pointer, thus child accessing with ->
	//###if(_CUDAFUNCS_TIMEIT)
	//##watch.lap("Cuda prelims: ");
	//###endif
	//##// Allocate space on device for device copies
	//##hipMalloc((void **)&xp_c,size_i);
	//##hipMalloc((void **)&yp_c,size_i);
	//##hipMalloc((void **)&wp_c,size_i);
	//##hipMalloc((void **)&mxp_c,size_i);
	//##hipMalloc((void **)&myp_c,size_i);
	//##hipMalloc((void **)&h_c,size_h);
	//###if(_CUDAFUNCS_TIMEIT)
	//##watch.lap("Allocate space on device: ");
	//###endif
	//##// Copy inputs to device
	//##hipMemcpy(xp_c,	xp,	size_i,	hipMemcpyHostToDevice);
	//##hipMemcpy(yp_c,	yp,	size_i,	hipMemcpyHostToDevice);
	//##hipMemcpy(wp_c,	wp,	size_i,	hipMemcpyHostToDevice);
	//##hipMemcpy(h_c,		hp,	size_h,	hipMemcpyHostToDevice);
	//###if(_CUDAFUNCS_TIMEIT)
	//##watch.lap("Copy mem host -> device: ");
	//###endif
	//##// Execute combine on cpu
	//##//std::cerr << "Execute device code." << std::endl;
	//##//calcmap_cuda<<<n_blocks,n_threads>>>(xp_c, yp_c, wp_c, mxp_c, myp_c, h_c);
	//##// Launch 2D grid
	//##// Source: http://www.informit.com/articles/article.aspx?p=2455391
	//##int TX = 32;
	//##int TY = 32;
	//##dim3 blockSize(TX, TY);
	//##//int bx = (wmax+ blockSize.x-1)/blockSize.x;
	//##//int by = (hmax+ blockSize.y-1)/blockSize.y;
	//##int bx = (wmax+ TX - 1)/TX;
	//##int by = (wmax+ TY - 1)/TY; // Correct? or hmax??
	//##dim3 gridSize = dim3 (bx, by);
	//##calcmap_cuda<<<gridSize, blockSize>>>(xp_c, yp_c, wp_c, mxp_c, myp_c, h_c, &wmax, &hmax);
	//###if(_CUDAFUNCS_TIMEIT)
	//##watch.lap("Execute device code: ");
	//###endif
	//##// copy results to host
	//##//std::cerr << "Copy memory from device to host." << std::endl;
	//##hipMemcpy(mxp, mxp_c, size_f, hipMemcpyDeviceToHost);
	//##hipMemcpy(myp, myp_c, size_f, hipMemcpyDeviceToHost);
	//###if(_CUDAFUNCS_TIMEIT)
	//##watch.lap("Copy mem device -> host: ");
	//###endif
	//##// cleanup device memory
	//##hipFree(mxp_c);	hipFree(myp_c),	hipFree(h_c);
	//##hipFree(xp_c);		hipFree(yp_c);		hipFree(wp_c);

	//###if(_CUDAFUNCS_DEBUG)
	//##std::cerr << "### calcmapping <end> ###" << std::endl;
	//###endif
	//##// Return nothing, void function.
	//##return;
}


// ######################################################################################
void copy(const cv::Mat& image_in, cv::Mat& image_out){
//	int device = 0;
//	SAFE_CALL(hipSetDevice(device),"CUDA Set Device Failed");
//	SAFE_CALL(hipFree(0),"CUDA Free Failed");
//	SAFE_CALL(hipDeviceSynchronize(),"CUDA Device Sync Failed");
//	SAFE_CALL(hipDeviceSynchronize(),"CUDA Thread Sync Failed");

	#if(_CUDAFUNCS_DEBUG)
	std::cerr << "### domapping <start> ###" << std::endl;
	#endif
	cv::imshow("image_input",	image_in);
	cv::imshow("image_out",		image_out);
	cv::waitKey(0);
	// calculate nubmer of bytes in input and output image
	const int inputBytes	= image_in.step*image_in.rows;
	const int outputBytes	= image_out.step*image_out.rows;
	unsigned char *d_input, *d_output;
	//std::cerr	<< "Rows input:           " << image_in.rows			<< std::endl;
	//std::cerr	<< "Cols input:           " << image_in.cols 			<< std::endl;
	//std::cerr	<< "Type input:           " << image_in.type()			<< std::endl;
	//std::cerr	<< "input continuous:     " << image_in.isContinuous()	<< std::endl;
	//std::cerr	<< "Step input:           " << image_in.step			<< std::endl;
	//std::cerr	<< "Rows image_out:       " << image_out.rows			<< std::endl;
	//std::cerr	<< "Cols image_out:       " << image_out.cols			<< std::endl;
	//std::cerr	<< "Type image_out:       " << image_out.type()			<< std::endl;
	//std::cerr	<< "image_out continuous: " << image_out.isContinuous()	<< std::endl;
	//std::cerr	<< "Step image_out:       " << image_out.step			<< std::endl;
	// Allocate device memory
	SAFE_CALL(hipMalloc<unsigned char>(&d_input,	inputBytes),	"CUDA Malloc input Failed");
	SAFE_CALL(hipMalloc<unsigned char>(&d_output,	outputBytes) ,	"CUDA Malloc output Failed");

	// Copy image_in to device
	SAFE_CALL(hipMemcpy(d_input, image_in.ptr(), inputBytes, hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");

	// Specify block size
	const dim3 block(16,16);
	// Calculate grid size to cover whole image
	const dim3 grid((image_in.cols + block.x-1)/block.x, (image_in.rows + block.y-1)/block.y);
	
	// Launch kernel
	copy_cuda<<<grid,block>>>(d_input,
							d_output,
							image_in.cols,
							image_out.cols,
							image_in.step,
							image_in.step);
	// ??TODO: see other types: http://horacio9573.no-ip.org/cuda/group__CUDART__MEMORY_g17f3a55e8c9aef5f90b67cdf22851375.html
	// Synchronize to check for kernel launch errors
	SAFE_CALL(hipDeviceSynchronize(),"Kernel Launch Failed");
	
	// Retrieve image_input from device
	SAFE_CALL(hipMemcpy(image_out.ptr(), d_output, outputBytes, hipMemcpyDeviceToHost), "CUDA Memcpy Device To Host Failed");
	
	// Free memory
	SAFE_CALL(hipFree(d_input) ,"CUDA Free Failed");
	SAFE_CALL(hipFree(d_output),"CUDA Free Failed");
}


// ######################################################################################
//void domapping(cv::Mat& image_output, const cv::Mat& image_input, Eigen::MatrixXf *Mx, Eigen::MatrixXf *My){
//// domapping
//// Function that performs the actual mapping
//// d_ stands for device	(gpu)
//// h_ stands for host	(cpu)
//	int device = 0;
//	SAFE_CALL(hipSetDevice(device),"CUDA Set Device Failed");
//	SAFE_CALL(hipFree(0),"CUDA Free Failed");
//	SAFE_CALL(hipDeviceSynchronize(),"CUDA Device Sync Failed");
//	SAFE_CALL(hipDeviceSynchronize(),"CUDA Thread Sync Failed");
//
//	const cv::Mat image_in = cv::imread("media/50x50.png",CV_LOAD_IMAGE_COLOR);
//	#if(_CUDAFUNCS_DEBUG)
//	std::cerr << "### domapping <start> ###" << std::endl;
//	#endif
//	cv::imshow("image_input",image_in);
//	cv::waitKey(0);
//	// create output image space on host
//	cv::Mat input_out(image_in.rows, image_in.cols, 16);
//	//const int input_bytes	= image_in.step*image_in.rows;
//	//const int output_bytes	= input_out.step*input_out.rows;
//	const int input_bytes	= image_in.rows*image_in.step  ;
//	const int output_bytes	= input_out.rows*input_out.step;
//	std::cerr	<< "Rows input:           " << image_in.rows				<< std::endl;
//	std::cerr	<< "Cols input:           " << image_in.cols 				<< std::endl;
//	std::cerr	<< "Type input:           " << image_in.type()			<< std::endl;
//	std::cerr	<< "input continuous:     " << image_in.isContinuous()	<< std::endl;
//	std::cerr	<< "Step input:           " << image_in.step				<< std::endl;
//	std::cerr	<< "Rows input_out:       " << input_out.rows				<< std::endl;
//	std::cerr	<< "Cols input_out:       " << input_out.cols				<< std::endl;
//	std::cerr	<< "Type input_out:       " << input_out.type()			<< std::endl;
//	std::cerr	<< "input_out continuous: " << input_out.isContinuous()	<< std::endl;
//	std::cerr	<< "Step input_out:       " << input_out.step				<< std::endl;
//	unsigned char *d_input, *d_output;
//	// TODO: load image here !! 
//	SAFE_CALL(hipMalloc<unsigned char>(&d_input,	input_bytes),	"CUDA Malloc input Failed");
//	SAFE_CALL(hipMalloc<unsigned char>(&d_output,	output_bytes) ,	"CUDA Malloc output Failed");
//	//SAFE_CALL(hipMalloc((void**)&d_input,	input_bytes) ,"CUDA Malloc input Failed");
//	//SAFE_CALL(hipMalloc((void**)&d_output,	output_bytes),"CUDA Malloc ouput Failed");
//
//	// Copy image_input to device
//	SAFE_CALL(hipMemcpy(d_input, image_in.ptr(), input_bytes, hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");
//
//	const dim3 block(16,16);
//	//const dim3 block(32,32);
//	//const dim3 block(64,64);
//	//const dim3 grid(ceil(float(image_in.cols + block.x-1)/float(block.x)), ceil(float(image_in.rows + block.y-1)/float(block.y)));
//	const dim3 grid(ceil((image_in.cols + block.x-1)/block.x), ceil((image_in.rows + block.y-1)/block.y));
//	// TODO: inspect grid
//	//std::cerr << ceil((image_in.cols + block.x-1)/block.x) <<std::endl;
//	//std::cerr << ceil((image_in.rows + block.y-1)/block.y) <<std::endl;
//	copy_cuda<<<grid,block>>>(d_input,
//							d_output,
//							image_in.step,
//							input_out.step,
//							image_in.cols,
//							image_in.rows);
//	SAFE_CALL(hipDeviceSynchronize(),"Kernel Launch Failed");
//	SAFE_CALL(hipDeviceSynchronize(),"CUDA Thread Sync Failed");
//	// TODO: see other types: http://horacio9573.no-ip.org/cuda/group__CUDART__MEMORY_g17f3a55e8c9aef5f90b67cdf22851375.html
//	// Retrieve image_input from device
//
//	SAFE_CALL(hipMemcpy(input_out.ptr(), d_output, output_bytes, hipMemcpyDeviceToHost), "CUDA Memcpy Device To Host Failed");
//	SAFE_CALL(hipDeviceSynchronize(),"CUDA Device Sync Failed");
//	SAFE_CALL(hipDeviceSynchronize(),"CUDA Thread Sync Failed");
//	// Free memory
//	SAFE_CALL(hipFree(d_input) ,"CUDA Free Failed");
//	SAFE_CALL(hipFree(d_output),"CUDA Free Failed");
//	std::cerr << "Show Image." << std::endl;
//	cv::imshow("image_output",input_out);
//	cv::waitKey(0);
//	std::cerr << "Image shown." << std::endl;
////	#if(_CUDAFUNCS_TIMEIT)
////	gputimer watch;
////	watch.start();
////	#endif
////	//cv::imshow("im_in",*image_in);
////	//cv::imshow("im_out",*image_out);
////	//cv::waitKey(0);
////	// TODO: upload input and output matrix to GPU
////	// TODO: inputmage dimensions and Mx and My not correpsonding when rotations and translations are zero.	 
////	int width_mx	= Mx->cols();
////	int height_mx	= Mx->rows();
////	int width_my	= My->cols();
////	int height_my	= My->rows();
////	int width_in	= image_in->cols;
////	int height_in	= image_in->rows;
////	int width_out	= image_out->cols;
////	int height_out	= image_out->rows;
////	int N_m			= width_mx*height_mx;
////	int N_in		= width_in*height_in;
////	int N_out		= width_out*height_out;
////	int channels	= image_in->channels();
////	// Determine size of memory for each input and output
////	int size_m	= N_m*sizeof(float);			// size of Mx and My (one channel)
////	int size_in	= N_in*sizeof(uchar)*channels;	// size of image_in	 (three channels)
////	int size_out= N_out*sizeof(uchar)*channels;	// size of image_out (three channels)
////	#if(_CUDAFUNCS_DEBUG)
////	std::cerr << "Width_mx:      " << width_mx		<< std::endl;
////	std::cerr << "Height_mx:     " << height_mx		<< std::endl;
////	std::cerr << "Width_my:      " << width_my		<< std::endl;
////	std::cerr << "Height_my:     " << height_my		<< std::endl;
////	std::cerr << "Width_in:      " << width_in		<< std::endl;
////	std::cerr << "Height_in:     " << height_in		<< std::endl;
////	std::cerr << "Width_out:     " << width_out		<< std::endl;
////	std::cerr << "Height_out:    " << height_out	<< std::endl;
////	std::cerr << "Channels:      " << channels		<< std::endl;
////	std::cerr << "size_m:        " << size_m		<< std::endl;
////	std::cerr << "size_in:       " << size_in		<< std::endl;
////	std::cerr << "size_out:      " << size_out		<< std::endl;
////	std::cerr << "sizeof(uchar): " << sizeof(uchar)	<< std::endl;
////	std::cerr << "sizeof(cv::CV_8U): " << sizeof(CV_8U)	<< std::endl;
////	std::cerr << "sizeof(float): " << sizeof(float)	<< std::endl;
////	std::cerr << "type image_in: " << image_in->type() << std::endl;
////	std::cerr << "type image_out:" << image_out->type() << std::endl;
////	#endif
//
//	// TODO, keep Mx and My on CUDA device?
//	// Create pointers
//	//float *mxp, *myp, *mxp_c, *myp_c;
//	//uchar *im_out_c, *im_in_c, *im_in, *im_out;
//	// Get pointers to data of mapping matrices
//	//mxp		= Mx->data();		// Mx is a pointer, thus child accessing with ->
//	//myp		= My->data();		// My is a pointer, thus child accessing with ->
//	////im_in	= image_in->data;	// Get pointer from cv::Mat
//	//im_out	= image_out->data;	// Get pointer fomr cv::Mat
//	////im_in	= image_in->ptr(0);	// Get pointer from cv::Mat
//	////im_out	= image_out->ptr(0);	// Get pointer fomr cv::Mat
//	//std::cerr << "sizeof(im_in): "	<< sizeof(im_in[0])	<< std::endl;
//	//std::cerr << "sizeof(im_out): "	<< sizeof(im_out[0])	<< std::endl;
//	
//	// Allocate space on device for device copies
//	//hipMalloc((void **)&mxp_c,		size_m);
//	//hipMalloc((void **)&myp_c,		size_m);
//	//hipMalloc((void **)&im_in_c,	size_in);
//	//hipMalloc((void **)&im_out_c,	size_out);
//	// Copy inputs to device
//	//hipMemcpy(mxp_c,	mxp,	size_m,		hipMemcpyHostToDevice);
//	//hipMemcpy(myp_c,	myp,	size_m,		hipMemcpyHostToDevice);
//	//hipMemcpy(im_in_c,	im_in,	size_in,	hipMemcpyHostToDevice);
////#	std::cerr << "Make GpuMat." << std::endl;
////#	cv::cuda::GpuMat image_in_c;
////#	std::cerr << "Upload Image." << std::endl;
////#	image_in_c.upload(*image_in);
//	//uchar *image_in_c;
//	//hipMalloc((void **)&im_in_c,	size_in);
//
//	//std::cerr << "Data host -> device." << std::endl;
//	//hipMemcpy2D(image_in_c.data, image_in_c.step, image_in->data, image_in->step, image_in->cols*image_in->elemSize(), image_in->rows,	hipMemcpyHostToDevice);
//
////	// Launch 2D grid
////	// Source: http://www.informit.com/articles/article.aspx?p=2455391
////	int TX = 32;
////	int TY = 32;
////	dim3 blockSize(TX, TY);
////	//int bx = (wmax+ blockSize.x-1)/blockSize.x;
////	//int by = (hmax+ blockSize.y-1)/blockSize.y;
////	int bx = (width_out+ TX - 1)/TX*channels;
//////	int by = (width_out+ TY - 1)/TY*channels;
////	int by = (height_out+ TY - 1)/TY*channels;
////	std::cerr << "bx: " << bx << ", by: " << by << std::endl;
////	dim3 gridSize = dim3 (bx, by);
////	//domap_cuda<<<gridSize, blockSize>>>(im_out_c, im_in_c, mxp_c, myp_c, &width_out, &height_out);
////	#if(_CUDAFUNCS_TIMEIT)
////	watch.lap("Execute mapping on device: ");
////	#endif
////
////	// TODO compare pointers to data
////	//std::cerr << "Pointer to host data:   " << im_in << std::endl;
////	//std::cerr << "Pointer to device data: " << im_in_c << std::endl;
////
////	std::cerr << "type image_in: " << image_in->type() << std::endl;
////	std::cerr << "Image in (zeros): "<<std::endl;
////	image_in->setTo(0);
////	for(int i = N_in*10; i < N_in; i++){
////		std::cerr << int(im_in[i]) << std::endl;
////	}
////	std::cerr << "type image_in: " << image_in->type() << std::endl;
////	// Get results back from host
////	//hipMemcpy(im_out,	im_out_c,	size_out,	hipMemcpyDeviceToHost);
////	//hipMemcpy(im_in,	im_in_c,	size_in,	hipMemcpyDeviceToHost);
////	std::cerr << "Step: " << step << std::endl;
////	//hipMemcpy2D(im_in, 0, im_in_c, step, sizeof(uchar)*width_in, sizeof(uchar)*height_in,	hipMemcpyDeviceToHost);
////#	std::cerr << "data device -> host." << std::endl;
////#	hipMemcpy2D(image_in->data, image_in->step, image_in_c.data, image_in_c.step, image_in_c.cols*image_in_c.elemSize(), image_in_c.rows,	hipMemcpyDeviceToHost);
////#
////#
////#	// TODO: this does not work.
////#	std::cerr << "Show images." << std::endl;
////#	cv::imshow("im_in",*image_in);	// correct
////#//	cv::imshow("im_out",*image_out);// incorrect
////#	cv::waitKey(0);
////#	// cleanup device memory
////#	//hipFree(mxp_c);	hipFree(myp_c);	hipFree(im_in_c); hipFree(im_out_c);
////#	//hipFree(im_in_c);
////#
//	#if(_CUDAFUNCS_TIMEIT)
//	watch.stop();
//	#endif
//	#if(_CUDAFUNCS_DEBUG)
//	std::cerr << "### domapping <end> ###" << std::endl;
//	#endif
//	// Return nothing, void function.
//	return;
//}
